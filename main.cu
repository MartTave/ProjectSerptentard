// Libraries
#include <iostream>
#include <sstream>
#include <string>
#include <sys/stat.h>
#include <hip/hip_runtime.h>

// == User lib ==
#include "diagnostics/diagnostics.h"
#include "initialization/init.h"
#include "solve/solve.h"
#include "write/write.h"

#include "common_includes.c"

// Namespace
using namespace std;

// Advection Solver
int main(int argc, char *argv[])
{

    // Data Initialization
    // == Spatial ==
    int scale = 1;
    if (argc > 1)
    {
        scale = stoi(argv[1]);
    }

    int nx = 100 * scale;
    int ny = 100 * scale; // Number of cells in each direction
    double Lx = 1.0;
    double Ly = 1.0; // Square domain [m]
    double dx = Lx / (nx - 1);
    double dy = Ly / (ny - 1); // Spatial step [m]

    // == Temporal ==
    double tFinal = 2.0;           // Final time [s]
    double dt = 0.005 / scale;     // Temporal step [s]
    int nSteps = int(tFinal / dt); // Number of steps to perform
    double time = 0.0;             // Actual Simulation time [s]

    // == Numerical ==
    int outputFrequency = nSteps / 40;

    float *h_phi;
    float *h_curvature;
    float *h_u;
    float *h_v;

    float *d_phi;
    float *d_phi_n;
    float *d_curvature;
    float *d_u;
    float *d_v;

    CHECK_ERROR(hipMalloc((void **)&d_phi, nx * ny));
    CHECK_ERROR(hipMalloc((void **)&d_phi_n, nx * ny));
    CHECK_ERROR(hipMalloc((void **)&d_curvature, nx * ny));
    CHECK_ERROR(hipMalloc((void **)&d_u, nx * ny));
    CHECK_ERROR(hipMalloc((void **)&d_v, nx * ny));

    Initialization(d_phi, d_curvature, d_u, d_v, nx, ny, dx, dy); // Initialize the distance function field
    computeBoundaries(d_phi, nx, ny);                             // Extrapolate phi on the boundaries
    cudaDeviceSyncronize();

    // == Output ==
    stringstream ss;
    ss << scale;
    string scaleStr = ss.str();

    string outputName = "output/levelSet_scale" + scaleStr + "_";
    int count = 0; // Number of VTK file already written

    // == First output ==
    // Write data in VTK format
    mkdir("output", 0777); // Create output folder

    // TODO: Memcopy from device to host
    writeDataVTK(outputName, phi, curvature, u, v, nx, ny, dx, dy, count++);

    // Loop over time
    for (int step = 1; step <= nSteps; step++)
    {

        time += dt; // Simulation time increases
        cout << "\nStarting iteration step " << step << "/" << nSteps << "\tTime " << time << "s\n";

        // Solve the advection equation
        solveAdvectionEquationExplicit(phi, u, v, nx, ny, dx, dy, dt);

        cudaDeviceSyncronize();

        // Diagnostics: interface perimeter
        computeInterfaceLength(phi, nx, ny, dx, dy);

        // Diagnostics: interface curvature
        computeInterfaceCurvature(phi, curvature, nx, ny, dx, dy);

        cudaDeviceSyncronize();

        // TODO: Memcopy from device to host (This time, no need to copy u and v)

        // Write data to output file
        if (step % outputFrequency == 0)
        {
            writeDataVTK(outputName, phi, curvature, u, v, nx, ny, dx, dy, count++);
        }
    }

    // Free memory
    delete[] h_phi, h_curvature, h_u, h_v;

    CHECK_ERROR(hipFree((void **)&d_phi));
    CHECK_ERROR(hipFree((void **)&d_phi_n));
    CHECK_ERROR(hipFree((void **)&d_curvature));
    CHECK_ERROR(hipFree((void **)&d_u));
    CHECK_ERROR(hipFree((void **)&d_v));

    return 0;
}
