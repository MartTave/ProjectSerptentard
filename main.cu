#include "hip/hip_runtime.h"
// Libraries
#include <iostream>
#include <sstream>
#include <string>
#include <sys/stat.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <chrono>

// == User lib ==
#include "diagnostics/diagnostics.cuh"
#include "initialization/init.cuh"
#include "solve/solve.cuh"
#include "write/write.h"

#include "common_includes.cu"

// Namespace
using namespace std;
using namespace std::chrono;

// Advection Solver
int main(int argc, char *argv[])
{
    MPI_Status status;

    int world_size, world_rank;

    MPI_Init(&argc, &argv);

    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    // Get the rank of the process
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);

    // Variables declaration
    int nx, ny, nSteps, scale, outputFrequency, gridWidth, gridHeight, windowSize;
    scale = 10;
    if (argc > 1)
    {
        scale = stoi(argv[1]);
    }
    nx = 100 * scale;
    ny = 100 * scale; // Number of cells in each direction

    long sum = 0;

    double Lx, Ly, dx, dy, tFinal, dt, time;

    long arrayLength, arraySplittedSize;
    arrayLength = nx * ny;
    stringstream ss;

    int count = 0; // Number of VTK file already written
    string scaleStr = ss.str();
    // == Output ==
    ss << scale;
    string outputName = "output/levelSet_scale" + scaleStr + "_";

    dim3 dimGrid, dimBlock;

    double *h_phi = new double[arrayLength];
    double *h_curvature = new double[arrayLength + (arrayLength % world_size)];
    double *h_u = new double[arrayLength + (arrayLength % world_size)];
    double *h_v = new double[arrayLength + (arrayLength % world_size)];
    double *h_lengths = new double[arrayLength + (arrayLength % world_size)];
    long size = arrayLength * sizeof(double);
    int *arrStart = new int[world_size];
    int *arrEnd = new int[world_size];
    int *splittedLengthes = new int[world_size];
    int *splittedSizes = new int[world_size];

    Lx = 1.0;
    Ly = 1.0; // Square domain [m]
    dx = Lx / (nx - 1);
    dy = Ly / (ny - 1); // Spatial step [m]

    // == Temporal ==
    tFinal = 4.0;              // Final time [s]
    dt = 0.005 / scale;        // Temporal step [s]
    nSteps = int(tFinal / dt); // Number of steps to perform
    time = 0.0;                // Actual Simulation time [s]

    // == Numerical ==
    outputFrequency = nSteps / 40;

    windowSize = 25;
    gridWidth = (nx + windowSize - 1) / windowSize;
    gridHeight = (ny + windowSize - 1) / windowSize;
    dimGrid = dim3(gridWidth, gridHeight);
    dimBlock = dim3(windowSize, windowSize);

    if (world_rank == 0)
    {
        int rest = arrayLength % world_size;
        int nbrOfElements = arrayLength / world_size;
        for (int i = 0; i < world_size; i++)
        {
            if (i < rest)
            {
                arrStart[i] = i * (nbrOfElements + 1);
                arrEnd[i] = (i + 1) * (nbrOfElements + 1);
                splittedLengthes[i] = (nbrOfElements + 1);
            }
            else
            {
                arrStart[i] = rest * (nbrOfElements + 1) + (i - rest) * nbrOfElements;
                arrEnd[i] = rest * (nbrOfElements + 1) + (i - rest + 1) * nbrOfElements;
                splittedLengthes[i] = nbrOfElements;
            }
            splittedSizes[i] = splittedLengthes[i] * sizeof(double);
            printf("Array start and end is : %d %d\n", arrStart[i], arrEnd[i]);
        }
    }

    MPI_Bcast(arrStart, world_size, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(arrEnd, world_size, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(splittedLengthes, world_size, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(splittedSizes, world_size, MPI_INT, 0, MPI_COMM_WORLD);

    double *h_phi_splitted = new double[splittedLengthes[world_rank]];
    double *h_curvature_splitted = new double[splittedLengthes[world_rank]];
    double *h_lengths_splitted = new double[splittedLengthes[world_rank]];
    double *h_u_splitted = new double[splittedLengthes[world_rank]];
    double *h_v_splitted = new double[splittedLengthes[world_rank]];

    double *d_phi;
    double *d_phi_n;
    double *d_curvature;
    double *d_lengths;
    double *d_u;
    double *d_v;

    if (world_rank == 0)
    {
        mkdir("output", 0777); // Create output folder
        CHECK_ERROR(hipMalloc((void **)&d_phi, size));
        CHECK_ERROR(hipMalloc((void **)&d_lengths, size));
        CHECK_ERROR(hipMalloc((void **)&d_phi_n, size));
        CHECK_ERROR(hipMalloc((void **)&d_curvature, size));
        CHECK_ERROR(hipMalloc((void **)&d_u, size));
        CHECK_ERROR(hipMalloc((void **)&d_v, size));

        InitializationKernel<<<dimGrid, dimBlock>>>(d_phi, d_curvature, d_u, d_v, nx, ny, dx, dy);
        hipDeviceSynchronize();
        computeBoundariesLines<<<1, nx>>>(d_phi, nx, ny);
        computeBoundariesColumns<<<1, ny>>>(d_phi, nx, ny);
        hipDeviceSynchronize();
    }

    size_t pointerSize = sizeof(void *);

    if (world_rank == 0)
    {
        CHECK_ERROR(hipMemcpy(h_phi, d_phi, size, hipMemcpyDeviceToHost));
        CHECK_ERROR(hipMemcpy(h_curvature, d_curvature, size, hipMemcpyDeviceToHost));
        CHECK_ERROR(hipMemcpy(h_u, d_u, size, hipMemcpyDeviceToHost));
        CHECK_ERROR(hipMemcpy(h_v, d_v, size, hipMemcpyDeviceToHost));
        for (int i = 1; i < world_size; i++)
        {
            MPI_Send(h_phi + arrStart[i], splittedLengthes[i], MPI_DOUBLE, i, 0, MPI_COMM_WORLD);
            MPI_Send(h_curvature + arrStart[i], splittedLengthes[i], MPI_DOUBLE, i, 0, MPI_COMM_WORLD);
            MPI_Send(h_u + arrStart[i], splittedLengthes[i], MPI_DOUBLE, i, 0, MPI_COMM_WORLD);
            MPI_Send(h_v + arrStart[i], splittedLengthes[i], MPI_DOUBLE, i, 0, MPI_COMM_WORLD);
        }

        // We can set the starting pointer to the first element of the array, as the size will delimit what will be sent to the next function
        h_phi_splitted = h_phi;
        h_curvature_splitted = h_curvature;
        h_u_splitted = h_u;
        h_v_splitted = h_v;
    }
    else
    {
        MPI_Recv(h_phi_splitted, splittedLengthes[world_rank], MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status);
        MPI_Recv(h_curvature_splitted, splittedLengthes[world_rank], MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status);
        MPI_Recv(h_u_splitted, splittedLengthes[world_rank], MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status);
        MPI_Recv(h_v_splitted, splittedLengthes[world_rank], MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status);
    }

    string toWriteU = getString(h_u_splitted, splittedLengthes[world_rank]);
    string toWriteV = getString(h_v_splitted, splittedLengthes[world_rank]);
    string toWritePhi = getString(h_phi_splitted, splittedLengthes[world_rank]);
    string toWriteCurvature = getString(h_curvature_splitted, splittedLengthes[world_rank]);
    if (world_rank == 0) {
	cout << "Writing initial data\n";
    }
    writeDataVTK(outputName, toWritePhi, toWriteCurvature, toWriteU, toWriteV, nx, ny, dx, dy, count++, world_rank);
    if (world_rank == 0) {
	cout << "Done - Written : " << outputName << "\n";
    }
    // Loop over time
    for (int step = 1; step <= nSteps; step++)
    {
        double max = 0;
        double total_length = 0;

        if (world_rank == 0)
        {

            time += dt; // Simulation time increases

            // Solve the advection equation
            copyPhi<<<dimGrid, dimBlock>>>(d_phi, d_phi_n, nx, ny);
            solveAdvectionEquationExplicit<<<dimGrid, dimBlock>>>(d_phi, d_phi_n, d_u, d_v, nx, ny, dx, dy, dt);

            hipDeviceSynchronize();

            computeBoundariesLines<<<1, nx>>>(d_phi, nx, ny);
            computeBoundariesColumns<<<1, ny>>>(d_phi, nx, ny);

            hipDeviceSynchronize();

            // Diagnostics: interface perimeter
            computeInterfaceLengthKernel<<<dimGrid, dimBlock>>>(d_phi, d_lengths, nx, ny, dx, dy);

            // Diagnostics: interface curvature
            computeInterfaceCurvatureKernel<<<dimGrid, dimBlock>>>(d_phi, d_curvature, nx, ny, dx, dy);

            hipDeviceSynchronize();
            CHECK_ERROR(hipMemcpy(h_phi, d_phi, size, hipMemcpyDeviceToHost));
            CHECK_ERROR(hipMemcpy(h_curvature, d_curvature, size, hipMemcpyDeviceToHost));
            CHECK_ERROR(hipMemcpy(h_lengths, d_lengths, size, hipMemcpyDeviceToHost));

            for (int i = 1; i < world_size; i++)
            {
                MPI_Send(h_phi + arrStart[i], splittedLengthes[i], MPI_DOUBLE, i, 0, MPI_COMM_WORLD);
                MPI_Send(h_curvature + arrStart[i], splittedLengthes[i], MPI_DOUBLE, i, 0, MPI_COMM_WORLD);
                MPI_Send(h_lengths + arrStart[i], splittedLengthes[i], MPI_DOUBLE, i, 0, MPI_COMM_WORLD);
            }
            // TODO:Maybe no need to do this, as the pointer still point to the same place
            h_phi_splitted = h_phi;
            h_curvature_splitted = h_curvature;
            h_lengths_splitted = h_lengths;
            cout << "Done loop\n";
        }
        else
        {
            MPI_Recv(h_phi_splitted, splittedLengthes[world_rank], MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status);
            MPI_Recv(h_curvature_splitted, splittedLengthes[world_rank], MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status);
            MPI_Recv(h_lengths_splitted, splittedLengthes[world_rank], MPI_DOUBLE, 0, 0, MPI_COMM_WORLD, &status);
        }

        double localSum = 0;
        double localMax = 0;
        for (int i = 0; i < arraySplittedSize; i++)
        {
            localSum += h_lengths_splitted[i];
            if (abs(h_curvature_splitted[i]) > localMax)
            {
                localMax = abs(h_curvature_splitted[i]);
            }
        }
        MPI_Reduce(&localMax, &max, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
        MPI_Reduce(&localSum, &total_length, 1, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);

        if (world_rank == 0)
        {
            cout << "Done reduce\n";
        }

        string toWritePhi = getString(h_phi_splitted, splittedLengthes[world_rank]);
        string toWriteCurvature = getString(h_curvature_splitted, splittedLengthes[world_rank]);

        writeDataVTK(outputName, toWritePhi, toWriteCurvature, toWriteU, toWriteU, nx, ny, dx, dy, count++, world_rank);

        // Write data to output file
        if (world_rank == 0 && step % outputFrequency == 0)
        {
            cout << "Step: " << step << "\n\n";
        }
    }

    delete[] h_phi, h_curvature, h_u, h_v;

    if (world_rank == 0)
    {
        // Free memory

        CHECK_ERROR(hipFree((void **)d_phi));
        CHECK_ERROR(hipFree((void **)d_phi_n));
        CHECK_ERROR(hipFree((void **)d_curvature));
        CHECK_ERROR(hipFree((void **)d_u));
        CHECK_ERROR(hipFree((void **)d_v));
        printf("File writing took : %ld ns\n", sum);
    }
    MPI_Finalize();
    return 0;
}
