#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include "../diagnostics/diagnostics.h"
#include "init.cuh"

using namespace std;


__global__ void InitializationKernel(double *phi, double *curvature, double *u, double *v, const int nx, const int ny, const double dx, const double dy)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // if (i >= nx || j >= ny) {
    //     printf("Killing index out of array bounds, with indexes : %d, %d\n", i, j);
    //     return;
    // }

    // == Circle parameters ==
    double xcenter = 0.5;  // Circle position x
    double ycenter = 0.75; // Circle position y
    double radius = 0.15;  // Circle radius

    double x = i * dx - xcenter;
    double y = j * dy - ycenter;

    // Compute the signed distance to the interface
    double distance = sqrt(x * x + y * y) - radius;

    phi[i * ny + j] = distance;

    // Compute the velocity based on x and y
    u[i * ny + j] = sin(2.0 * M_PI * j * dy) * sin(M_PI * i * dx) * sin(M_PI * i * dx);
    v[i * ny + j] = -sin(2.0 * M_PI * i * dx) * sin(M_PI * j * dy) * sin(M_PI * j * dy);
}